#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime_api.h"
#include "sz_opencl_kernels.h"

__global__ void
calculate_regression_coefficents_kernel(
        const cl_float* oriData,
        struct sz_opencl_sizes const* sizes,
        float* reg_params,
        float* const pred_buffer)
{
    unsigned long i = threadIdx.x + blockIdx.x * blockDim.x;//get_global_id(0);
    unsigned long j = threadIdx.y + blockIdx.y * blockDim.y;//get_global_id(1);
		unsigned long k = threadIdx.z + blockIdx.z * blockDim.z;//get_global_id(2);
      const unsigned int block_id =
        i * (sizes->num_y * sizes->num_z) + j * sizes->num_z + k;
			if(block_id < sizes->num_blocks) {
      const float* cur_data_pos = pred_buffer + (block_id * sizes->max_num_block_elements);
      float fx = 0.0;
      float fy = 0.0;
      float fz = 0.0;
      float f = 0;
      float sum_x, sum_y;
      float curData;
      for (size_t i = 0; i < sizes->block_size; i++) {
        sum_x = 0;
        for (size_t j = 0; j < sizes->block_size; j++) {
          sum_y = 0;
          for (size_t k = 0; k < sizes->block_size; k++) {
            curData = *cur_data_pos;
            sum_y += curData;
            fz += curData * k;
            cur_data_pos++;
          }
          fy += sum_y * j;
          sum_x += sum_y;
        }
        fx += sum_x * i;
        f += sum_x;
      }
      float coeff =
        1.0 / (sizes->block_size * sizes->block_size * sizes->block_size);
      float* reg_params_pos = reg_params + block_id;
      reg_params_pos[0] = (2 * fx / (sizes->block_size - 1) - f) * 6 * coeff /
                          (sizes->block_size + 1);
      reg_params_pos[sizes->params_offset_b] =
        (2 * fy / (sizes->block_size - 1) - f) * 6 * coeff /
        (sizes->block_size + 1);
      reg_params_pos[sizes->params_offset_c] =
        (2 * fz / (sizes->block_size - 1) - f) * 6 * coeff /
        (sizes->block_size + 1);
      reg_params_pos[sizes->params_offset_d] =
        f * coeff -
        ((sizes->block_size - 1) * reg_params_pos[0] / 2 +
         (sizes->block_size - 1) * reg_params_pos[sizes->params_offset_b] / 2 +
         (sizes->block_size - 1) * reg_params_pos[sizes->params_offset_c] / 2);
		}
}

#define CUDA_SAFE_CALL(call) {                                    \
  hipError_t err = call;                                                    \
  if( hipSuccess != err) {                                                \
    fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
        __FILE__, __LINE__, hipGetErrorString( err) );              \
    exit(EXIT_FAILURE);                                                  \
  } \
}


#define CUDA_SAFE_KERNEL_CALL(call) {                                    \
	call; \
	hipError_t err = hipGetLastError(); \
  if( hipSuccess != err) {                                                \
    fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
        __FILE__, __LINE__, hipGetErrorString( err) );              \
    exit(EXIT_FAILURE);                                                  \
  } \
}

void
calculate_regression_coefficents_host(
        const cl_float* oriData,
        struct sz_opencl_sizes const* sizes,
        float* reg_params,
        float* const pred_buffer){
  int deviceNum;
  unsigned int maxBlockSize;
  hipGetDevice(&deviceNum);
  hipDeviceGetAttribute((int*)&maxBlockSize, hipDeviceAttributeMaxThreadsPerBlock, deviceNum);
  maxBlockSize = floor(cbrt(maxBlockSize));


  dim3 block_size{maxBlockSize,maxBlockSize,maxBlockSize};
  dim3 grid_size{sizes->num_x/maxBlockSize + 1, sizes->num_y/maxBlockSize + 1, sizes->num_z/maxBlockSize + 1};

  float* oriData_d;
  struct sz_opencl_sizes* sizes_d;
  float* reg_params_d;
  float* pred_buffer_d;

  CUDA_SAFE_CALL(hipMalloc(&oriData_d, sizeof(cl_float) * sizes->num_elements));
  CUDA_SAFE_CALL(hipMalloc(&sizes_d, sizeof(sz_opencl_sizes)));
  CUDA_SAFE_CALL(hipMalloc(&reg_params_d, sizeof(cl_float) * sizes->reg_params_buffer_size));
  CUDA_SAFE_CALL(hipMalloc(&pred_buffer_d, sizeof(cl_float) * sizes->num_blocks * sizes->max_num_block_elements));

  CUDA_SAFE_CALL(hipMemcpy(oriData_d, oriData, sizeof(cl_float) * sizes->num_elements, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(sizes_d, sizes, sizeof(struct sz_opencl_sizes), hipMemcpyHostToDevice));

  CUDA_SAFE_KERNEL_CALL((calculate_regression_coefficents_kernel<<<grid_size, block_size>>>(oriData_d, sizes_d, reg_params_d, pred_buffer_d)));

  CUDA_SAFE_CALL(hipMemcpy(reg_params, reg_params_d, sizeof(cl_float) * sizes->reg_params_buffer_size, hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipMemcpy(pred_buffer, pred_buffer_d, sizeof(cl_float) * sizes->num_blocks * sizes->max_num_block_elements, hipMemcpyDeviceToHost));

  CUDA_SAFE_CALL(hipFree(oriData_d));
  CUDA_SAFE_CALL(hipFree(sizes_d));
  CUDA_SAFE_CALL(hipFree(reg_params_d));
  CUDA_SAFE_CALL(hipFree(pred_buffer_d));
}

__global__
void copy_block_data_kernel(
    float* data,
    sz_opencl_decompress_positions const* pos,
    float const * dec_block_data
    ) {

  unsigned long i = threadIdx.x + blockIdx.x * blockDim.x;//get_global_id(0);
  unsigned long j = threadIdx.y + blockIdx.y * blockDim.y;//get_global_id(1);
  if(i < pos->data_elms1 && j < pos->data_elms2) {
    const float *block_data_pos =
        dec_block_data + (i + pos->resi_x) * pos->dec_block_dim0_offset + (j + pos->resi_y) * pos->dec_block_dim1_offset
            + pos->resi_z;
    float *final_data_pos = data + i * pos->data_elms2 * pos->data_elms3 + j * pos->data_elms3;
    for (cl_ulong k = 0; k < pos->data_elms3; k++) {
      *(final_data_pos++) = *(block_data_pos++);
    }
  }
}


void copy_block_data_host(float **data,
                     const sz_opencl_decompress_positions &pos,
                     const float *dec_block_data) {// extract data
  *data = (float*)malloc(sizeof(cl_float) * pos.data_buffer_size);

  float* data_d;
  float* dec_block_data_d;
  sz_opencl_decompress_positions* pos_d;

  int deviceNum;
  unsigned int maxBlockSize2;
  hipGetDevice(&deviceNum);
  hipDeviceGetAttribute((int*)&maxBlockSize2, hipDeviceAttributeMaxThreadsPerBlock, deviceNum);
  maxBlockSize2 = floor(sqrt(maxBlockSize2));


  dim3 block_size(maxBlockSize2,maxBlockSize2);
  dim3 grid_size(pos.data_elms1/maxBlockSize2 + 1, pos.data_elms2/maxBlockSize2 + 1);

  CUDA_SAFE_CALL(hipMalloc(&data_d, sizeof(cl_float)* pos.data_buffer_size));
  CUDA_SAFE_CALL(hipMalloc(&dec_block_data_d, sizeof(cl_float) *pos.dec_block_data_size));
  CUDA_SAFE_CALL(hipMalloc(&pos_d, sizeof(sz_opencl_decompress_positions)));

  CUDA_SAFE_CALL(hipMemcpy(pos_d, &pos, sizeof(struct sz_opencl_decompress_positions), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(dec_block_data_d, dec_block_data, sizeof(cl_float)* pos.dec_block_data_size, hipMemcpyHostToDevice));
  //do not copy data since we just malloc'ed it

  CUDA_SAFE_KERNEL_CALL((copy_block_data_kernel<<<grid_size,block_size>>>(data_d, pos_d, dec_block_data_d)));

  CUDA_SAFE_CALL(hipMemcpy(*data, data_d, sizeof(cl_float) * pos.data_buffer_size, hipMemcpyDeviceToHost));
  //do not copy sizes_d or pos_d, or dec_block_data_d back because they are const

  CUDA_SAFE_CALL(hipFree(data_d));
  CUDA_SAFE_CALL(hipFree(pos_d));
  CUDA_SAFE_CALL(hipFree(dec_block_data_d));

}
