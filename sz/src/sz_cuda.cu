#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include <limits>
#include <stdexcept>
#include "hip/hip_runtime_api.h"
#include "sz_opencl_kernels.h"

#define CUDA_SAFE_CALL(call) {                                    \
  hipError_t err = call;                                                    \
  if( hipSuccess != err) {                                                \
    fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
        __FILE__, __LINE__, hipGetErrorString( err) );              \
    exit(EXIT_FAILURE);                                                  \
  } \
}


#define CUDA_SAFE_KERNEL_CALL(call) {                                    \
	call; \
	hipError_t err = hipGetLastError(); \
  if( hipSuccess != err) {                                                \
    fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
        __FILE__, __LINE__, hipGetErrorString( err) );              \
    exit(EXIT_FAILURE);                                                  \
  } \
}

template <class  T, class U>
auto integer_divide_up(T a, U b) {
  size_t val = (a % b != 0) ? (a/b+1) : (a/b);
  if(val > std::numeric_limits<T>::max())
    throw std::domain_error("invalid integer division");
  else return val;
}


__global__ void
calculate_regression_coefficents_kernel(
        const cl_float* oriData,
        struct sz_opencl_sizes const* sizes,
        float* reg_params,
        float* const pred_buffer)
{
    unsigned long i = threadIdx.x + blockIdx.x * blockDim.x;//get_global_id(0);
    unsigned long j = threadIdx.y + blockIdx.y * blockDim.y;//get_global_id(1);
		unsigned long k = threadIdx.z + blockIdx.z * blockDim.z;//get_global_id(2);
      const unsigned int block_id =
        i * (sizes->num_y * sizes->num_z) + j * sizes->num_z + k;
			if(block_id < sizes->num_blocks) {
      const float* cur_data_pos = pred_buffer + (block_id * sizes->max_num_block_elements);
      float fx = 0.0;
      float fy = 0.0;
      float fz = 0.0;
      float f = 0;
      float sum_x, sum_y;
      float curData;
      for (size_t i = 0; i < sizes->block_size; i++) {
        sum_x = 0;
        for (size_t j = 0; j < sizes->block_size; j++) {
          sum_y = 0;
          for (size_t k = 0; k < sizes->block_size; k++) {
            curData = *cur_data_pos;
            sum_y += curData;
            fz += curData * k;
            cur_data_pos++;
          }
          fy += sum_y * j;
          sum_x += sum_y;
        }
        fx += sum_x * i;
        f += sum_x;
      }
      float coeff =
        1.0 / (sizes->block_size * sizes->block_size * sizes->block_size);
      float* reg_params_pos = reg_params + block_id;
      reg_params_pos[0] = (2 * fx / (sizes->block_size - 1) - f) * 6 * coeff /
                          (sizes->block_size + 1);
      reg_params_pos[sizes->params_offset_b] =
        (2 * fy / (sizes->block_size - 1) - f) * 6 * coeff /
        (sizes->block_size + 1);
      reg_params_pos[sizes->params_offset_c] =
        (2 * fz / (sizes->block_size - 1) - f) * 6 * coeff /
        (sizes->block_size + 1);
      reg_params_pos[sizes->params_offset_d] =
        f * coeff -
        ((sizes->block_size - 1) * reg_params_pos[0] / 2 +
         (sizes->block_size - 1) * reg_params_pos[sizes->params_offset_b] / 2 +
         (sizes->block_size - 1) * reg_params_pos[sizes->params_offset_c] / 2);
		}
}

void
calculate_regression_coefficents_host(
        const cl_float* oriData,
        struct sz_opencl_sizes const* sizes,
        float* reg_params,
        float* const pred_buffer){
  int deviceNum;
  unsigned int maxBlockSize;
  hipGetDevice(&deviceNum);
  hipDeviceGetAttribute((int*)&maxBlockSize, hipDeviceAttributeMaxThreadsPerBlock, deviceNum);
  maxBlockSize = floor(cbrt(maxBlockSize));


  dim3 block_size{maxBlockSize,maxBlockSize,maxBlockSize};
  dim3 grid_size{sizes->num_x/maxBlockSize + 1, sizes->num_y/maxBlockSize + 1, sizes->num_z/maxBlockSize + 1};

  float* oriData_d;
  struct sz_opencl_sizes* sizes_d;
  float* reg_params_d;
  float* pred_buffer_d;

  CUDA_SAFE_CALL(hipMalloc(&oriData_d, sizeof(cl_float) * sizes->num_elements));
  CUDA_SAFE_CALL(hipMalloc(&sizes_d, sizeof(sz_opencl_sizes)));
  CUDA_SAFE_CALL(hipMalloc(&reg_params_d, sizeof(cl_float) * sizes->reg_params_buffer_size));
  CUDA_SAFE_CALL(hipMalloc(&pred_buffer_d, sizeof(cl_float) * sizes->data_buffer_size));

  CUDA_SAFE_CALL(hipMemcpy(oriData_d, oriData, sizeof(cl_float) * sizes->num_elements, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(sizes_d, sizes, sizeof(struct sz_opencl_sizes), hipMemcpyHostToDevice));

  CUDA_SAFE_KERNEL_CALL((calculate_regression_coefficents_kernel<<<grid_size, block_size>>>(oriData_d, sizes_d, reg_params_d, pred_buffer_d)));

  CUDA_SAFE_CALL(hipMemcpy(reg_params, reg_params_d, sizeof(cl_float) * sizes->reg_params_buffer_size, hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipMemcpy(pred_buffer, pred_buffer_d, sizeof(cl_float) * sizes->data_buffer_size, hipMemcpyDeviceToHost));

  CUDA_SAFE_CALL(hipFree(oriData_d));
  CUDA_SAFE_CALL(hipFree(sizes_d));
  CUDA_SAFE_CALL(hipFree(reg_params_d));
  CUDA_SAFE_CALL(hipFree(pred_buffer_d));
}

__global__
void copy_block_data_kernel(
    float* data,
    sz_opencl_decompress_positions const* pos,
    float const * dec_block_data
    ) {

  unsigned long i = threadIdx.x + blockIdx.x * blockDim.x;//get_global_id(0);
  unsigned long j = threadIdx.y + blockIdx.y * blockDim.y;//get_global_id(1);
  if(i < pos->data_elms1 && j < pos->data_elms2) {
    const float *block_data_pos =
        dec_block_data + (i + pos->resi_x) * pos->dec_block_dim0_offset + (j + pos->resi_y) * pos->dec_block_dim1_offset
            + pos->resi_z;
    float *final_data_pos = data + i * pos->data_elms2 * pos->data_elms3 + j * pos->data_elms3;
    for (cl_ulong k = 0; k < pos->data_elms3; k++) {
      *(final_data_pos++) = *(block_data_pos++);
    }
  }
}


void copy_block_data_host(float **data,
                     const sz_opencl_decompress_positions &pos,
                     const float *dec_block_data) {// extract data
  *data = (float*)malloc(sizeof(cl_float) * pos.data_buffer_size);

  float* data_d;
  float* dec_block_data_d;
  sz_opencl_decompress_positions* pos_d;

  int deviceNum;
  unsigned int maxBlockSize2;
  hipGetDevice(&deviceNum);
  hipDeviceGetAttribute((int*)&maxBlockSize2, hipDeviceAttributeMaxThreadsPerBlock, deviceNum);
  maxBlockSize2 = floor(sqrt(maxBlockSize2));


  dim3 block_size(maxBlockSize2,maxBlockSize2);
  dim3 grid_size(pos.data_elms1/maxBlockSize2+1, pos.data_elms2/maxBlockSize2+1);

  CUDA_SAFE_CALL(hipMalloc(&data_d, sizeof(cl_float)* pos.data_buffer_size));
  CUDA_SAFE_CALL(hipMalloc(&dec_block_data_d, sizeof(cl_float) *pos.dec_block_data_size));
  CUDA_SAFE_CALL(hipMalloc(&pos_d, sizeof(sz_opencl_decompress_positions)));

  CUDA_SAFE_CALL(hipMemcpy(pos_d, &pos, sizeof(struct sz_opencl_decompress_positions), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(dec_block_data_d, dec_block_data, sizeof(cl_float)* pos.dec_block_data_size, hipMemcpyHostToDevice));
  //do not copy data since we just malloc'ed it

  CUDA_SAFE_KERNEL_CALL((copy_block_data_kernel<<<grid_size,block_size>>>(data_d, pos_d, dec_block_data_d)));

  CUDA_SAFE_CALL(hipMemcpy(*data, data_d, sizeof(cl_float) * pos.data_buffer_size, hipMemcpyDeviceToHost));
  //do not copy sizes_d or pos_d, or dec_block_data_d back because they are const

  CUDA_SAFE_CALL(hipFree(data_d));
  CUDA_SAFE_CALL(hipFree(pos_d));
  CUDA_SAFE_CALL(hipFree(dec_block_data_d));

}

__global__
void prepare_data_buffer_kernel(const float *oriData, const sz_opencl_sizes *sizes, cl_float *data_buffer) {
  unsigned long i = threadIdx.x + blockIdx.x * blockDim.x;//get_global_id(0);
  unsigned long j = threadIdx.y + blockIdx.y * blockDim.y;//get_global_id(1);
  unsigned long k = threadIdx.z + blockIdx.z * blockDim.z;//get_global_id(2);
  unsigned int block_id = i * (sizes->num_y * sizes->num_z) + j * sizes->num_z + k;

  if(block_id < sizes->num_blocks) {
    cl_float *data_buffer_location = data_buffer + block_id * sizes->max_num_block_elements;
    for (unsigned int ii = 0; ii < sizes->block_size; ii++) {
      for (unsigned int jj = 0; jj < sizes->block_size; jj++) {
        for (unsigned int kk = 0; kk < sizes->block_size; kk++) {
          // index in origin data
          cl_ulong i_ = i * sizes->block_size + ii;
          cl_ulong j_ = j * sizes->block_size + jj;
          cl_ulong k_ = k * sizes->block_size + kk;
          i_ = (i_ < sizes->r1) ? i_ : sizes->r1 - 1;
          j_ = (j_ < sizes->r2) ? j_ : sizes->r2 - 1;
          k_ = (k_ < sizes->r3) ? k_ : sizes->r3 - 1;
          data_buffer_location[ii * sizes->block_size * sizes->block_size + jj * sizes->block_size + kk] =
              oriData[i_ * sizes->r2 * sizes->r3 + j_ * sizes->r3 + k_];
        }
      }
    }
  }
}


void prepare_data_buffer_host(float const *oriData, sz_opencl_sizes const *sizes, cl_float *data_buffer) {
  float* oriData_d;
  sz_opencl_sizes* sizes_d;
  float* data_buffer_d;

  int deviceNum;
  unsigned int maxBlockSize;
  hipGetDevice(&deviceNum);
  hipDeviceGetAttribute((int*)&maxBlockSize, hipDeviceAttributeMaxThreadsPerBlock, deviceNum);
  maxBlockSize = floor(cbrt(maxBlockSize));


  dim3 block_size{maxBlockSize,maxBlockSize,maxBlockSize};
  dim3 grid_size(integer_divide_up(sizes->num_x,maxBlockSize), integer_divide_up(sizes->num_y,maxBlockSize), integer_divide_up(sizes->num_z, maxBlockSize));


  CUDA_SAFE_CALL(hipMalloc(&oriData_d, sizeof(cl_float) * sizes->num_elements));
  CUDA_SAFE_CALL(hipMalloc(&sizes_d, sizeof(sz_opencl_sizes)));
  CUDA_SAFE_CALL(hipMalloc(&data_buffer_d, sizeof(cl_float) * sizes->num_blocks * sizes->max_num_block_elements));

  CUDA_SAFE_CALL(hipMemcpy(oriData_d, oriData, sizeof(cl_float) * sizes->num_elements, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(sizes_d, sizes, sizeof(struct sz_opencl_sizes), hipMemcpyHostToDevice));
  //do not copy data buffer since it is created in this function

  CUDA_SAFE_KERNEL_CALL((prepare_data_buffer_kernel<<<grid_size, block_size>>>(oriData_d, sizes_d, data_buffer_d)));

  CUDA_SAFE_CALL(hipMemcpy(data_buffer, data_buffer_d, sizeof(cl_float) * sizes->num_blocks * sizes->max_num_block_elements, hipMemcpyDeviceToHost));
  //do not copy sizes_d or oriData_d since they are const copied

  CUDA_SAFE_CALL(hipFree(oriData_d));
  CUDA_SAFE_CALL(hipFree(sizes_d));
  CUDA_SAFE_CALL(hipFree(data_buffer_d));
}

__device__
void
compute_errors_kernel(const float* reg_params_pos, const float* data_buffer,
               sz_opencl_sizes const* sizes, float mean, float noise,
               bool use_mean, size_t i, size_t j, size_t k,
               float& err_sz, float& err_reg)
{
  const float* cur_data_pos =
      data_buffer +
          i * sizes->block_size * sizes->block_size +
          j * sizes->block_size + k;
  float curData = *cur_data_pos;
  float pred_sz =
      cur_data_pos[-1] + cur_data_pos[-sizes->strip_dim1_offset] +
          cur_data_pos[-sizes->strip_dim0_offset] -
          cur_data_pos[-sizes->strip_dim1_offset - 1] -
          cur_data_pos[-sizes->strip_dim0_offset - 1] -
          cur_data_pos[-sizes->strip_dim0_offset - sizes->strip_dim1_offset] +
          cur_data_pos[-sizes->strip_dim0_offset - sizes->strip_dim1_offset - 1];
  float pred_reg = reg_params_pos[0] * i +
      reg_params_pos[sizes->params_offset_b] * j +
      reg_params_pos[sizes->params_offset_c] * k +
      reg_params_pos[sizes->params_offset_d];
  if (use_mean) {
    err_sz += min(fabs(pred_sz - curData) + noise, fabs(mean - curData));
    err_reg += fabs(pred_reg - curData);
  } else {
    err_sz += fabs(pred_sz - curData) + noise;
    err_reg += fabs(pred_reg - curData);
  }
}


__global__
void
opencl_sample_kernel(const sz_opencl_sizes* sizes,
                     float mean,
                     float noise,
                     bool use_mean,
                     const float* data_buffer,
                     const float* reg_params_pos,
                     unsigned char* indicator_pos
) {
  unsigned long i = threadIdx.x + blockIdx.x * blockDim.x;//get_global_id(0);
  unsigned long j = threadIdx.y + blockIdx.y * blockDim.y;//get_global_id(1);
  unsigned long k = threadIdx.z + blockIdx.z * blockDim.z;//get_global_id(2);
  const unsigned int block_id = i * (sizes->num_y * sizes->num_z) + j * sizes->num_z + k;
  if(block_id < sizes->num_blocks) {
    const float *data_pos = data_buffer + (block_id * sizes->max_num_block_elements);
    /*sampling and decide which predictor*/
    {
      // sample point [1, 1, 1] [1, 1, 4] [1, 4, 1] [1, 4, 4] [4, 1, 1] [4,
      // 1, 4] [4, 4, 1] [4, 4, 4]
      float err_sz = 0.0, err_reg = 0.0;
      for (size_t block_i = 1; block_i < sizes->block_size; block_i++) {
        int bmi = sizes->block_size - block_i;
        compute_errors_kernel(&reg_params_pos[block_id], data_pos, sizes, mean, noise,
                              use_mean, block_i, block_i, block_i, err_sz,
                              err_reg);
        compute_errors_kernel(&reg_params_pos[block_id], data_pos, sizes, mean, noise,
                              use_mean, block_i, block_i, bmi, err_sz, err_reg);

        compute_errors_kernel(&reg_params_pos[block_id], data_pos, sizes, mean, noise,
                              use_mean, block_i, bmi, block_i, err_sz, err_reg);

        compute_errors_kernel(&reg_params_pos[block_id], data_pos, sizes, mean, noise,
                              use_mean, block_i, bmi, bmi, err_sz, err_reg);
      }
      indicator_pos[(i * sizes->num_y + j) * sizes->num_z + k] = err_reg >= err_sz;
    }
  }
}

void
opencl_sample_host(const sz_opencl_sizes* sizes,
              float mean,
              float noise,
              bool use_mean,
              const float* data_buffer,
              const float* reg_params_pos,
              unsigned char* indicator_pos
)
{
  int deviceNum;
  unsigned int maxBlockSize;
  hipGetDevice(&deviceNum);
  hipDeviceGetAttribute((int*)&maxBlockSize, hipDeviceAttributeMaxThreadsPerBlock, deviceNum);
  maxBlockSize = floor(cbrt(maxBlockSize));

  dim3 block_size{maxBlockSize,maxBlockSize,maxBlockSize};
  dim3 grid_size(integer_divide_up(sizes->num_x,maxBlockSize), integer_divide_up(sizes->num_y,maxBlockSize), integer_divide_up(sizes->num_z, maxBlockSize));

  sz_opencl_sizes* sizes_d;
  float* data_buffer_d;
  float* reg_params_pos_d;
  unsigned char* indicator_pos_d;

  CUDA_SAFE_CALL(hipMalloc(&data_buffer_d, sizeof(cl_float) * sizes->data_buffer_size));
  CUDA_SAFE_CALL(hipMalloc(&sizes_d, sizeof(sz_opencl_sizes)));
  CUDA_SAFE_CALL(hipMalloc(&reg_params_pos_d, sizeof(cl_float) * sizes->reg_params_buffer_size));
  CUDA_SAFE_CALL(hipMalloc(&indicator_pos_d, sizes->num_blocks * sizeof(unsigned char)));

  CUDA_SAFE_CALL(hipMemcpy(data_buffer_d, data_buffer, sizeof(cl_float) * sizes->data_buffer_size, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(sizes_d, sizes, sizeof(sz_opencl_sizes), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(reg_params_pos_d, reg_params_pos, sizeof(cl_float) * sizes->reg_params_buffer_size, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(indicator_pos_d, indicator_pos, sizes->num_blocks * sizeof(unsigned char), hipMemcpyHostToDevice));

  CUDA_SAFE_KERNEL_CALL((opencl_sample_kernel<<<grid_size,block_size>>>(sizes_d, mean, noise, use_mean, data_buffer_d, reg_params_pos_d, indicator_pos_d)));

  CUDA_SAFE_CALL(hipMemcpy(indicator_pos, indicator_pos_d, sizes->num_blocks * sizeof(unsigned char), hipMemcpyDeviceToHost));

  CUDA_SAFE_CALL(hipFree(data_buffer_d));
  CUDA_SAFE_CALL(hipFree(sizes_d));
  CUDA_SAFE_CALL(hipFree(reg_params_pos_d));
  CUDA_SAFE_CALL(hipFree(indicator_pos_d));


}
